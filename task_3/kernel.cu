#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdio.h>
#include <time.h>
#include <random>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include ""
#include <chrono>

#define MAX 6 //maximum of your dice
#define MIN 1 //minimum of your dice
#define Blocks 4 * 32 * 32 * 32 //Number of blocks, for changing the sample size easily

__global__ void setup_kernel(hiprandState* state) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(1234, index, 0, &state[index]);
}

__global__ void monte_carlo_kernel(hiprandState* state, int* count, int m) {
	unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;

	__shared__ int cache[256];
	cache[threadIdx.x] = 0;
	__syncthreads();

	unsigned int temp = 0;
	int sumroll = 0;
	while (temp < m) {
		sumroll += int(((hiprand_uniform(&state[index])) * (MAX - MIN + 0.999999)) + MIN);
		temp++;
	} if (sumroll == 3 * m) {
		cache[threadIdx.x] = 1;
	}

	// reduction
	int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i) {
			cache[threadIdx.x] += cache[threadIdx.x + i];
		}

		i /= 2;
		__syncthreads();
	}


	// update to our global variable count
	if (threadIdx.x == 0) {
		atomicAdd(count, cache[0]);
	}

}

int main()
{
	unsigned int n = Blocks * 256; //Number of runs, If you change this, change the first number and change BLOCKS accordingly
	unsigned int m = 3; //Number of dice
	int* h_count;
	int* d_count;
	hiprandState* d_state;
	float chance;


	// allocate memory
	h_count = (int*)malloc(n * sizeof(int));
	hipMalloc((void**)&d_count, n * sizeof(int));
	hipMalloc((void**)&d_state, n * sizeof(hiprandState));
	hipMemset(d_count, 0, sizeof(int));

	//Taking start time
	auto start = std::chrono::steady_clock::now();

	// set kernel
	dim3 BLOCKS = Blocks;
	dim3 THREADS = 256;
	setup_kernel << < BLOCKS, THREADS >> > (d_state);


	// monte carlo kernel
	monte_carlo_kernel << <BLOCKS, THREADS >> > (d_state, d_count, m);

	//Taking the end time
	auto end = std::chrono::steady_clock::now();
	double time_elapsed_ns = double(std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count());

	// copy results back to the host
	hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

	// display results for gpu
	std::cout << *h_count << std::endl;
	chance = float(*h_count) / float(n);
	std::cout << "Chance is " << chance << std::endl;
	std::cout << "Time taken to run: " << time_elapsed_ns / 1e6 << std::endl;


	// delete memory
	free(h_count);
	hipFree(d_count);
	hipFree(d_state);
}