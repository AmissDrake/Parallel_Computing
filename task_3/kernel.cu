#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdio.h>
#include <time.h>
#include <random>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include ""

#define MAX 6 //maximum of your dice
#define MIN 1 //minimum of your dice


__global__ void setup_kernel(hiprandState* state) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(1234, index, 0, &state[index]);
}

__global__ void monte_carlo_kernel(hiprandState* state, int* count, int m) {
	unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;

	__shared__ int cache[256];
	cache[threadIdx.x] = 0;
	__syncthreads();

	unsigned int temp = 0;
	int sumroll = 0;
	while (temp < m) {
		sumroll += int(((hiprand_uniform(&state[index])) * (MAX - MIN + 0.999999)) + MIN);
		temp++;
	} if (sumroll == 3 * m) {
		cache[threadIdx.x] = 1;
	}

	// reduction
	int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i) {
			cache[threadIdx.x] += cache[threadIdx.x + i];
		}

		i /= 2;
		__syncthreads();
	}


	// update to our global variable count
	if (threadIdx.x == 0) {
		atomicAdd(count, cache[0]);
	}

}

int main()
{
	unsigned int n = 1024 * 4 * 256; //Number of runs,If you change this, you must also change THREADS
	unsigned int m = 2; //Number of dice
	int* h_count;
	int* d_count;
	hiprandState* d_state;
	float chance;


	// allocate memory
	h_count = (int*)malloc(n * sizeof(int));
	hipMalloc((void**)&d_count, n * sizeof(int));
	hipMalloc((void**)&d_state, n * sizeof(hiprandState));
	hipMemset(d_count, 0, sizeof(int));

	// set kernel
	dim3 BLOCKS = 1024*4;
	dim3 THREADS = 256;
	setup_kernel << < BLOCKS, THREADS >> > (d_state);


	// monte carlo kernel
	monte_carlo_kernel << <BLOCKS, THREADS >> > (d_state, d_count, m);


	// copy results back to the host
	hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);


	// display results for gpu
	std::cout << *h_count << std::endl;
	chance = float(*h_count) / float(n);
	std::cout << "Chance is " << chance << std::endl;


	// delete memory
	free(h_count);
	hipFree(d_count);
	hipFree(d_state);
}
