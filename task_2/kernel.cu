﻿#include "hip/hip_runtime.h"


#include <iostream>
#include <list>
#include <vector>

using std::cout; using std::vector;

vector<vector<int>> components; //Global declaration of components vector

//Graph representation using adjacency matrix
class Graph {
    public:
    int Vert;
    //Creating our adjacency matrix with name Adj
    vector<vector<bool>>Adj;

    //Init 
    Graph(int Vert){
        this->Vert=Vert;
        //Actually making the adjacency matrix
        Adj.resize(Vert,vector<bool>(Vert,false));
        }

    //Method(?) to add an edge to the graph
    void AddEdge(int x, int y){
        Adj[x][y]=true;
        Adj[y][x]=true; //Assuming undirected graph.
    }

    //Method to number of connected components
    int findComponents(){
    bool visited[Vert];
    int count;
    for(int i=0; i<Vert;i++){
        visited[i] = false;
    }
    for(int i=0; i<Vert;i++){
        if (!visited[i]){
            vector<int> subcomponents = {};
            dfs(i, visited, subcomponents);
            components.push_back(subcomponents);
            count++;
        }
    }
    return count;
    }

    //Defining a dfs function
    void dfs(int at, bool* visited, vector<int> &subcomponents){
    visited[at] = true;
    subcomponents.push_back(at);
    for(int nbridx = 0; nbridx<Adj[at].size(); nbridx++){
        if(!visited[Adj[at][nbridx]]){
            dfs(Adj[at][nbridx], visited, subcomponents);
        }
    }
    return;
    }

    //Method to print the graph
    void PrintGraph(){
        for(int i=0; i<Vert;i++){
            cout<<i<<"\t";
            for(int j=0; j<Adj[i].size();j++){
                cout<<Adj[i][j]<<",";
            }
            cout<<std::endl;
        }
    }
};