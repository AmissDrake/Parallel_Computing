﻿#include "hip/hip_runtime.h"


#include <iostream>
#include <cstdlib> //For randomizing input array
#include <chrono> //For measuring runtime
#include <cmath>
 
__global__ void ArrayAdd(int* A, int* B, int* C, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		C[i] = A[i] + B[i];
	}
	return;
}

void arrayinit(int* A, const int N) {
	for (int i = 0; i < N; i++) {
		A[i] = i;
	}
}

int main() {
	//Taking start time	
	auto start = std::chrono::steady_clock::now();

	const int N = 2 << 15; //Edit the array size here

	//Initializing arrays
	int A[N];
	int B[N];
	int C[N];
	arrayinit(A, N);
	arrayinit(B, N);

	//Use this codeblock if you want to set custom array values
	//int A[N] = { 1,2,3,4,5 };
	//int B[N] = { 1,2,3,4,5 };
	//int C[N] = {};

	//Creating GPU pointers
	int* gpuA = 0;
	int* gpuB = 0;
	int* gpuC = 0;

	//Allocating memory in the GPU
	hipMalloc(&gpuA, sizeof(A));
	hipMalloc(&gpuB, sizeof(B));	
	hipMalloc(&gpuC, sizeof(C));

	//Copying the arrays into the GPU memory
	hipMemcpy(gpuA, A, sizeof(A), hipMemcpyHostToDevice);	
	hipMemcpy(gpuB, B, sizeof(B), hipMemcpyHostToDevice);

	auto startactual = std::chrono::steady_clock::now();
	//Blocks and threads
	int THREADS = 1024;
	int BLOCKS = (N + THREADS - 1)/THREADS;
	//Calling the function
	ArrayAdd <<<BLOCKS,THREADS >>> (gpuA, gpuB, gpuC, N);
	//cudaDeviceSynchronize();
	auto endactual = std::chrono::steady_clock::now();

	//Copying the sum back to main memory
	hipMemcpy(C, gpuC, sizeof(C), hipMemcpyDeviceToHost);

	//Taking the end time
	auto end = std::chrono::steady_clock::now();
	//Measuring time difference
	double time_elapsed_ns = double(std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count());
	double actual_time_elapsed_ns = double(std::chrono::duration_cast<std::chrono::nanoseconds>(endactual - startactual).count());
	std::cout<<"total time elapsed: " << time_elapsed_ns / 1e6 << std::endl;
	std::cout <<"actual time elapsed: " << actual_time_elapsed_ns / 1e6 << std::endl;

	return 0;
}